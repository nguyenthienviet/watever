#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>
#include <math.h>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		y[i] = a*x[i] + y[i];
		if (threadIdx.x == 0) printf("y[%d] = %f\n", i, y[i]);
	}
}

int main(void)
{
	int N = 1 << 20;
	float *x, *y, *d_x, *d_y;

	x = (float*)malloc(N * sizeof(float));
	y = (float*)malloc(N * sizeof(float));

	hipMalloc(&d_x, N * sizeof(float));
	hipMalloc(&d_y, N * sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

	saxpy << <(N + 255) / 256, 256 >> >(N, 2.0f, d_x, d_y);
	hipDeviceSynchronize();

	hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = max(maxError, abs(y[i] - 4.0f));
	printf("Max error: %f\n", maxError);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

	return 0;
}